// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

//********************************************************//
// CUDA SIFT extractor by Marten Bjorkman aka Celebrandil //
//********************************************************//

#include <cstdio>

#include "cudaImage.h"
#include "cudautils.h"

int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }
int iDivDown(int a, int b) { return a / b; }
int iAlignUp(int a, int b) { return (a % b != 0) ? (a - a % b + b) : a; }
int iAlignDown(int a, int b) { return a - a % b; }

void CudaImage::Allocate(
        int w, int h, int p, bool host, float *devmem, float *hostmem) {
    width = w;
    height = h;
    pitch = p;
    d_data = devmem;
    h_data = hostmem;
    t_data = NULL;
    if (devmem == NULL) {
        safeCall(hipMallocPitch((void **)&d_data, (size_t *)&pitch,
                                 (size_t)(sizeof(float) * width),
                                 (size_t)height));
        pitch /= sizeof(float);
        if (d_data == NULL) printf("Failed to allocate device data\n");
        d_internalAlloc = true;
    }
    if (host && hostmem == NULL) {
        h_data = (float *)malloc(sizeof(float) * pitch * height);
        h_internalAlloc = true;
    }
}

CudaImage::CudaImage()
    : width(0),
      height(0),
      h_data(NULL),
      d_data(NULL),
      t_data(NULL),
      d_internalAlloc(false),
      h_internalAlloc(false) {}

CudaImage::~CudaImage() {
    if (d_internalAlloc && d_data != NULL) safeCall(hipFree(d_data));
    d_data = NULL;
    if (h_internalAlloc && h_data != NULL) free(h_data);
    h_data = NULL;
    if (t_data != NULL) safeCall(hipFreeArray((hipArray *)t_data));
    t_data = NULL;
}

double CudaImage::Download() {
    TimerGPU timer(0);
    int p = sizeof(float) * pitch;
    if (d_data != NULL && h_data != NULL)
        safeCall(hipMemcpy2D(d_data, p, h_data, sizeof(float) * width,
                              sizeof(float) * width, height,
                              hipMemcpyHostToDevice));
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("Download time =               %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}

double CudaImage::Readback() {
    TimerGPU timer(0);
    int p = sizeof(float) * pitch;
    safeCall(hipMemcpy2D(h_data, sizeof(float) * width, d_data, p,
                          sizeof(float) * width, height,
                          hipMemcpyDeviceToHost));
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("Readback time =               %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}

double CudaImage::InitTexture() {
    TimerGPU timer(0);
    hipChannelFormatDesc t_desc = hipCreateChannelDesc<float>();
    safeCall(hipMallocArray((hipArray **)&t_data, &t_desc, pitch, height));
    if (t_data == NULL) printf("Failed to allocated texture data\n");
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("InitTexture time =            %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}

double CudaImage::CopyToTexture(CudaImage &dst, bool host) {
    if (dst.t_data == NULL) {
        printf("Error CopyToTexture: No texture data\n");
        return 0.0;
    }
    if ((!host || h_data == NULL) && (host || d_data == NULL)) {
        printf("Error CopyToTexture: No source data\n");
        return 0.0;
    }
    TimerGPU timer(0);
    if (host)
        safeCall(hipMemcpy2DToArray(
                (hipArray *)dst.t_data, 0, 0, h_data, sizeof(float) * pitch,
                sizeof(float) * pitch, dst.height, hipMemcpyHostToDevice));
    else
        safeCall(hipMemcpy2DToArray(
                (hipArray *)dst.t_data, 0, 0, d_data, sizeof(float) * pitch,
                sizeof(float) * pitch, dst.height, hipMemcpyDeviceToDevice));
    safeCall(hipDeviceSynchronize());
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("CopyToTexture time =          %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}
